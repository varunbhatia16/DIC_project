#include "hip/hip_runtime.h"
#include "overlapTestConsts.h"

// matrix multiplication code taken from the CUDA SDK

__global__ void
matrixMul(float* C, float* A, float* B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void cudaMatMul(int matrixSize, ElementType *A, ElementType *B, ElementType *C) {
  hipStream_t matMulStream; 
  hipStreamCreate(&matMulStream); 
  ElementType *h_A, *h_B, *h_C; 
  ElementType *d_A, *d_B, *d_C;
  int size = matrixSize * matrixSize * sizeof(ElementType);

  hipHostMalloc((void **) &h_A, size); 
  hipHostMalloc((void **) &h_B, size); 
  hipHostMalloc((void **) &h_C, size);  

  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  memcpy(h_A, A, size);
  memcpy(h_B, B, size); 

  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, matMulStream); 
  hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, matMulStream); 

  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(matrixSize / threads.x, matrixSize / threads.y);
  
  // execute the kernel
  matrixMul<<< grid, threads, 0, matMulStream >>>(d_C, d_A, d_B, matrixSize, matrixSize);  

  hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, matMulStream); 

  hipStreamSynchronize(matMulStream); 

  memcpy(C, h_C, size);

  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_C);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipStreamDestroy(matMulStream); 
}
